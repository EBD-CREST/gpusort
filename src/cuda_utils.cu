#include "hip/hip_runtime.h"
#include "cuda/cuda_utils.h"

#include <algorithm>
#include <functional>
#include <iostream>
#include <inttypes.h>
#include <iterator>
#include <stdint.h>
#include <typeinfo>

#include <omp.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>

#include "common.h"
#include "cuda/cuda_custom_type.h"
#include "omp/omp_utils.h"

#ifdef __DO_BENCHMARK
#include "timer.h"
#endif

namespace gpusort {

//
// Global timers.
//
#ifdef __DO_BENCHMARK
Timer local_sort_mem_getinfo_tm;
Timer local_sort_merge_tm;
Timer local_sort_sort_tm;
Timer local_sort_transfer_tm;
#endif

//
// Declare internal functions.
//
template<typename K>
static inline thrust::less<K> Convert2ThrustComp(std::less<K> comp) {
  return thrust::less<K>();
}

template<typename K>
static inline thrust::greater<K> Convert2ThrustComp(std::greater<K> comp) {
  return thrust::greater<K>();
}

template<typename StrictWeakOrdering, typename DifferenceType,
         typename RandomAccessIterator>
__host__ void ParallelMergeArrays(StrictWeakOrdering comp,
                                  int num_threads, int num_chunks,
                                  const DifferenceType *displs,
                                  RandomAccessIterator in,
                                  RandomAccessIterator out);

void RaiseError(hipError_t err) throw(ThrustException);

void SelectDevice(int rank) throw(ThrustException);

template<typename StrictWeakOrdering, typename RandomAccessIterator>
void SortByThrust(StrictWeakOrdering comp,
                  RandomAccessIterator first,
                  RandomAccessIterator last,
                  RandomAccessIterator out) throw(ThrustException);

template<typename StrictWeakOrdering, typename RandomAccessIterator1,
         typename RandomAccessIterator2, typename KeyValuePair>
void SortByThrust(StrictWeakOrdering comp,
                  RandomAccessIterator1 key_first,
                  RandomAccessIterator1 key_last,
                  RandomAccessIterator2 val_first,
                  KeyValuePair *out) throw(ThrustException);

void SplitChunks(int64_t num_elems, int64_t size, int64_t *chunk_size,
                 int *num_chunks) throw(ThrustException);

//
// Implement Common API.
//
int CudaUtils::GetDeviceCount() throw(ThrustException) {
  int num_dev = 0;
  hipError_t cuda_state = hipGetDeviceCount(&num_dev);
  if (cuda_state != hipSuccess) {
    RaiseError(cuda_state);
  }
  return num_dev;
}

bool CudaUtils::IsHavingDevices() throw(ThrustException) {
  return (GetDeviceCount() > 0);
}

//
// Implement Sort API.
//
template<typename StrictWeakOrdering, typename RandomAccessIterator>
__host__ void CudaUtils::Sort(
    StrictWeakOrdering comp, int rank, RandomAccessIterator first,
    RandomAccessIterator last) throw(ThrustException) {
  typedef typename std::iterator_traits<RandomAccessIterator>::difference_type
      DiffType;
  typedef typename std::iterator_traits<RandomAccessIterator>::value_type
      ValueType;

#ifdef __DEBUG_MSG
  if (rank == 0) std::cout << "start CudaUtils::Sort" << std::endl;
#endif
  int64_t size = last - first, chunk_size = 0;
  int num_chunks = 0;
  SelectDevice(rank);
  SplitChunks(size, sizeof(ValueType), &chunk_size, &num_chunks);
  int tree_height = ceil(log(num_chunks) / log(2));

  if (num_chunks == 1) {
    SortByThrust(comp, first, last, first);
  } else {
    int64_t remainder = size % chunk_size;
    RandomAccessIterator buffer = new ValueType[size];
    int64_t *displs = new DiffType[num_chunks+1];
    displs[0] = 0;
    displs[num_chunks] = size;

    // sort part
    RandomAccessIterator tmp_first = first;
    RandomAccessIterator tmp_last = first + chunk_size;
    RandomAccessIterator tmp_buf_first = buffer;
    RandomAccessIterator tmp_buf_last = buffer + chunk_size;
    for (int i = 0; i < num_chunks; i++) {
      displs[i+1] = displs[i] + chunk_size;
      if (i == num_chunks-2 && remainder != 0) {
        chunk_size = remainder;
      }
      try {
        if (tree_height % 2 == 0) {
          SortByThrust(comp, tmp_first, tmp_last, tmp_first);
        } else {
          SortByThrust(comp, tmp_first, tmp_last, tmp_buf_first);
        }
      } catch (ThrustException &e) {
        delete[] buffer;
        delete[] displs;
        throw e;
      }
      tmp_first = tmp_last;
      tmp_last = tmp_first + chunk_size;
      tmp_buf_first = tmp_buf_last;
      tmp_buf_last = tmp_buf_first + chunk_size;
    }

    int max_threads = OmpUtils::GetMaxThreads();
  #ifdef __DO_BENCHMARK
    local_sort_merge_tm.Start();
  #endif
    if (tree_height % 2 == 0) {
      ParallelMergeArrays(comp, max_threads, num_chunks, displs, first, buffer);
    } else {
      ParallelMergeArrays(comp, max_threads, num_chunks, displs, buffer, first);
    }
  #ifdef __DO_BENCHMARK
    local_sort_merge_tm.Stop();
  #endif

    delete[] buffer;
    delete[] displs;
  }
}

template<typename StrictWeakOrdering, typename RandomAccessIterator1,
         typename RandomAccessIterator2>
__host__ void CudaUtils::SortByKey(
    StrictWeakOrdering comp, int rank,
    RandomAccessIterator1 key_first, RandomAccessIterator1 key_last,
    RandomAccessIterator2 val_first,
    void *_out) throw(ThrustException) {
  typedef typename std::iterator_traits<RandomAccessIterator1>::difference_type
      DiffType;
  typedef typename std::iterator_traits<RandomAccessIterator1>::value_type
      KeyType;
  typedef typename std::iterator_traits<RandomAccessIterator2>::value_type
      ValueType;
  typedef std::pair<KeyType, ValueType> KeyValuePair;

  KeyValuePair *out = reinterpret_cast<KeyValuePair*>(_out);
  int64_t size = key_last - key_first, chunk_size = 0;
  int num_chunks = 0;
  SelectDevice(rank);
  SplitChunks(size, sizeof(ValueType) + sizeof(KeyType),
              &chunk_size, &num_chunks);
  int tree_height = ceil(log(num_chunks) / log(2));

  if (num_chunks == 1) {
    SortByThrust(comp, key_first, key_last, val_first, out);
  } else {
    int64_t remainder = size % chunk_size;
    KeyValuePair *buffer = new KeyValuePair[size];
    int64_t *displs = new DiffType[num_chunks+1];
    displs[0] = 0;
    displs[num_chunks] = size;

    // sort part
    int64_t d1 = 0;
    int64_t d2 = d1 + chunk_size;
    for (int i = 0; i < num_chunks; i++) {
      try {
        if (tree_height % 2 == 0) {
          SortByThrust(comp, key_first + d1, key_first + d2,
                       val_first + d1, out + d1);
        } else {
          SortByThrust(comp, key_first + d1, key_first + d2,
                       val_first + d1, buffer + d1);
        }
      } catch (ThrustException &e) {
        delete[] buffer;
        delete[] displs;
        throw e;
      }
      if (i < num_chunks - 1)
        displs[i+1] = displs[i] + chunk_size;

      d1 += chunk_size;
      d2 = d1 + ((i == num_chunks-2 && remainder != 0)? remainder : chunk_size);
    }

    int max_threads = OmpUtils::GetMaxThreads();
  #ifdef __DO_BENCHMARK
    local_sort_merge_tm.Start();
  #endif
    if (tree_height % 2 == 0) {
      ParallelMergeArrays(ConvertComp<KeyValuePair>(comp),
                          max_threads, num_chunks, displs, out, buffer);
    } else {
      ParallelMergeArrays(ConvertComp<KeyValuePair>(comp),
                          max_threads, num_chunks, displs, buffer, out);
    }
  #ifdef __DO_BENCHMARK
    local_sort_merge_tm.Stop();
  #endif

    delete[] buffer;
    delete[] displs;
  }
}

//
// Pre-define template parameters for Sort function
// to pass the compiler linkage.
//
#define DECLARE_SORT_TEMPLATE(key_t)\
template void CudaUtils::Sort<std::less<key_t>, key_t*>(\
    std::less<key_t> comp, int rank,\
    key_t* first, key_t* last) throw(ThrustException);\
template void CudaUtils::Sort<std::greater<key_t>, key_t*>(\
    std::greater<key_t> comp, int rank,\
    key_t* first, key_t* last) throw(ThrustException);

DECLARE_SORT_TEMPLATE(int);  // For primitive type: int.
DECLARE_SORT_TEMPLATE(unsigned int);  // For primitive type: unsigned int.
DECLARE_SORT_TEMPLATE(float);  // For primitive type: float.
DECLARE_SORT_TEMPLATE(double);  // For primitive type: double.
DECLARE_SORT_TEMPLATE(int64_t);  // For primitive type: int64_t.
DECLARE_SORT_TEMPLATE(uint64_t);  // For primitive type: uint64_t.
DECLARE_SORT_TEMPLATE(CudaCustomType<int>);  // For custom data type
                                             // (key type is int).
DECLARE_SORT_TEMPLATE(CudaCustomType<unsigned int>);  // For custom data type
                                                      // (key type is uint).
DECLARE_SORT_TEMPLATE(CudaCustomType<float>);  // For custom data type
                                               // (key type is float).
DECLARE_SORT_TEMPLATE(CudaCustomType<double>);  // For custom data type
                                                // (key type is double).
DECLARE_SORT_TEMPLATE(CudaCustomType<int64_t>);  // For custom data type
                                                 // (key type is int64_t).
DECLARE_SORT_TEMPLATE(CudaCustomType<uint64_t>);  // For custom data type
                                                  // (key type is uint64_t).

//
// Pre-define template parameters for SortByKey function
// to pass the compiler linkage.
//
#define DECLARE_SORT_BY_KEY_TEMPLATE(key_t, value_t)\
template void CudaUtils::SortByKey<std::less<key_t>, key_t*, value_t*>(\
    std::less<key_t> comp, int rank, key_t* k_first, key_t* k_last,\
    value_t* v_first, void *_out) throw(ThrustException);\
template void CudaUtils::SortByKey<std::greater<key_t>, key_t*, value_t*>(\
    std::greater<key_t> comp, int rank, key_t* k_first, key_t* k_last,\
    value_t* v_first, void *_out) throw(ThrustException);

DECLARE_SORT_BY_KEY_TEMPLATE(int, int);
DECLARE_SORT_BY_KEY_TEMPLATE(unsigned int, unsigned int);
DECLARE_SORT_BY_KEY_TEMPLATE(float, float);
DECLARE_SORT_BY_KEY_TEMPLATE(double, double);
DECLARE_SORT_BY_KEY_TEMPLATE(int64_t, int64_t);
DECLARE_SORT_BY_KEY_TEMPLATE(uint64_t, uint64_t);

#define DECLARE_SORT_BY_KEY_PAIR_TEMPLATE(type1, type2)\
  DECLARE_SORT_BY_KEY_TEMPLATE(type1, type2);\
  DECLARE_SORT_BY_KEY_TEMPLATE(type2, type1);

// int pair with the remainders.
DECLARE_SORT_BY_KEY_PAIR_TEMPLATE(int, unsigned int);
DECLARE_SORT_BY_KEY_PAIR_TEMPLATE(int, float);
DECLARE_SORT_BY_KEY_PAIR_TEMPLATE(int, double);
DECLARE_SORT_BY_KEY_PAIR_TEMPLATE(int, int64_t);
DECLARE_SORT_BY_KEY_PAIR_TEMPLATE(int, uint64_t);
// unsigned int pair with the remainders.
DECLARE_SORT_BY_KEY_PAIR_TEMPLATE(unsigned int, float);
DECLARE_SORT_BY_KEY_PAIR_TEMPLATE(unsigned int, double);
DECLARE_SORT_BY_KEY_PAIR_TEMPLATE(unsigned int, int64_t);
DECLARE_SORT_BY_KEY_PAIR_TEMPLATE(unsigned int, uint64_t);
// float pair with the remainders.
DECLARE_SORT_BY_KEY_PAIR_TEMPLATE(float, double);
DECLARE_SORT_BY_KEY_PAIR_TEMPLATE(float, int64_t);
DECLARE_SORT_BY_KEY_PAIR_TEMPLATE(float, uint64_t);
// double pair with the remainders.
DECLARE_SORT_BY_KEY_PAIR_TEMPLATE(double, int64_t);
DECLARE_SORT_BY_KEY_PAIR_TEMPLATE(double, uint64_t);
// int64_t pair with the remainders.
DECLARE_SORT_BY_KEY_PAIR_TEMPLATE(int64_t, uint64_t);

//
//  Implement internal functions.
//
template<typename StrictWeakOrdering, typename DifferenceType,
         typename RandomAccessIterator>
__host__ void ParallelMergeArrays(StrictWeakOrdering comp,
                                  int num_threads, int num_chunks,
                                  const DifferenceType *displs,
                                  RandomAccessIterator in,
                                  RandomAccessIterator out) {
  int current_height = 0;
  int num_merge = (num_chunks + 2 - 1) / 2;
  int stride = 1;
  while (num_merge > 0) {
    RandomAccessIterator first1, last1, first2, last2, first3;
    for (int i = 0; i < num_merge; i++) {
      int offset_idx = i * stride * 2;
      int idx_first1 = (offset_idx > num_chunks)? num_chunks : offset_idx;
      int idx_last1 = offset_idx + stride;
      idx_last1 = (idx_last1 > num_chunks)? num_chunks : idx_last1;
      int idx_first2 = offset_idx + stride;
      idx_first2 = (idx_first2 > num_chunks)? num_chunks : idx_first2;
      int idx_last2 = offset_idx + (2 * stride);
      idx_last2 = (idx_last2 > num_chunks)? num_chunks : idx_last2;
      int idx_first3 = offset_idx;

      if (current_height % 2 == 0) {
        first1 = &in[displs[idx_first1]];
        last1 = &in[displs[idx_last1]];
        first2 = &in[displs[idx_first2]];
        last2 = &in[displs[idx_last2]];
        first3 = &out[displs[idx_first3]];
      } else if (current_height % 2 == 1) {
        first1 = &out[displs[idx_first1]];
        last1 = &out[displs[idx_last1]];
        first2 = &out[displs[idx_first2]];
        last2 = &out[displs[idx_last2]];
        first3 = &in[displs[idx_first3]];
      }

      OmpUtils::Merge(comp, first1, last1, first2, last2, first3);
    }

    if (num_merge == 1) {
      num_merge = 0;
    } else {
      num_merge = (num_merge + 2 - 1) / 2;
      stride = stride * 2;
      current_height++;
    }
  }
}

void RaiseError(hipError_t err) throw(ThrustException) {
  switch (err) {
    case hipErrorContextAlreadyInUse:
      throw ThrustException("A call tried to access an exclusive-thread device"
                            " that is already in use by a different thread");

    case hipErrorInsufficientDriver:
      throw ThrustException("The installed CUDA driver is older than"
                            " the CUDA runtime library");

    case hipErrorInvalidDevice:
      throw ThrustException("The device ordinal supplied by the user does not"
                            " correspond to a valid CUDA device");

    case hipErrorNoDevice:
      throw ThrustException("No CUDA-capable devices were detected");

    default:
      throw ThrustException("Unknown error occurred");
  }
}

void SelectDevice(int rank) throw(ThrustException) {
  int num_dev = CudaUtils::GetDeviceCount();
  hipError_t cuda_state = hipSetDevice((rank + 1) % num_dev);
  if (cuda_state != hipSuccess) {
    RaiseError(cuda_state);
  }
}

template<typename StrictWeakOrdering, typename RandomAccessIterator>
void SortByThrust(StrictWeakOrdering comp,
                  RandomAccessIterator first, RandomAccessIterator last,
                  RandomAccessIterator out) throw(ThrustException) {
  typedef typename std::iterator_traits<RandomAccessIterator>::value_type
      ValueType;
#ifdef __DO_BENCHMARK
  local_sort_transfer_tm.Start();
#endif
  thrust::device_vector<ValueType> d_v(0);
  try {
    d_v.assign(first, last);
  } catch(std::bad_alloc &e) {
    throw ThrustException("Couldn't allocate device vector");
  }
#ifdef __DO_BENCHMARK
  local_sort_transfer_tm.Stop();

  local_sort_sort_tm.Start();
#endif
  try {
    thrust::sort(d_v.begin(), d_v.end(), Convert2ThrustComp(comp));
  } catch (std::bad_alloc &e) {
    throw ThrustException("Ran out of memory while sorting");
  } catch (thrust::system_error &e) {
    throw ThrustException("Some other error happened during sort: "
                          + std::string(e.what()));
  }
#ifdef __DO_BENCHMARK
  local_sort_sort_tm.Stop();

  local_sort_transfer_tm.Start();
#endif
  try {
    thrust::copy(d_v.begin(), d_v.end(), out);
  } catch (thrust::system_error &e) {
    throw ThrustException("Some other error happened during copy: "
                          + std::string(e.what()));
  }
#ifdef __DO_BENCHMARK
  local_sort_transfer_tm.Stop();
#endif
}

template<typename StrictWeakOrdering, typename RandomAccessIterator1,
         typename RandomAccessIterator2, typename KeyValuePair>
void SortByThrust(StrictWeakOrdering comp,
                  RandomAccessIterator1 key_first,
                  RandomAccessIterator1 key_last,
                  RandomAccessIterator2 val_first,
                  KeyValuePair *out) throw(ThrustException) {
  typedef typename std::iterator_traits<RandomAccessIterator1>::value_type
      KeyType;
  typedef typename std::iterator_traits<RandomAccessIterator2>::value_type
      ValueType;

  int64_t nelem = key_last - key_first;
#ifdef __DO_BENCHMARK
  local_sort_transfer_tm.Start();
#endif
  thrust::device_vector<KeyType> d_k(0);
  thrust::device_vector<ValueType> d_v(0);
  try {
    d_k.assign(key_first, key_last);
    d_v.assign(val_first, val_first + nelem);
  } catch(std::bad_alloc &e) {
    throw ThrustException("Couldn't allocate device vector");
  }
#ifdef __DO_BENCHMARK
  local_sort_transfer_tm.Stop();

  local_sort_sort_tm.Start();
#endif
  try {
    thrust::sort_by_key(d_k.begin(), d_k.end(), d_v.begin(),
                        Convert2ThrustComp(comp));
  } catch (std::bad_alloc &e) {
    throw ThrustException("Ran out of memory while sorting");
  } catch (thrust::system_error &e) {
    throw ThrustException("Some other error happened during sort: "
                          + std::string(e.what()));
  }
#ifdef __DO_BENCHMARK
  local_sort_sort_tm.Stop();

  local_sort_transfer_tm.Start();
#endif
  try {
    thrust::copy(d_k.begin(), d_k.end(), key_first);
    thrust::copy(d_v.begin(), d_v.end(), val_first);
  } catch (thrust::system_error &e) {
    throw ThrustException("Some other error happened during copy: "
                          + std::string(e.what()));
  }

  int n_threads = OmpUtils::GetMaxThreads();
  FOR_PARALLEL(n_threads, nelem, i,
               {
                 out[i] = std::make_pair(key_first[i], val_first[i]);
               });
#ifdef __DO_BENCHMARK
  local_sort_transfer_tm.Stop();
#endif
}

void SplitChunks(int64_t num_elems, int64_t size, int64_t *chunk_size,
                 int *num_chunks) throw(ThrustException) {
  size_t mem_avai = 1, mem_total = 1;
  local_sort_mem_getinfo_tm.Start();
  hipError_t cuda_state = hipMemGetInfo(&mem_avai, &mem_total);
  local_sort_mem_getinfo_tm.Stop();
  if (cuda_state != hipSuccess) RaiseError(cuda_state);
  int64_t chunk_size_ = mem_avai / (3 * size);
  int num_chunks_ = (num_elems + chunk_size_ - 1) / chunk_size_;

  *chunk_size = chunk_size_;
  *num_chunks = num_chunks_;
}

}  // namespace gpusort
